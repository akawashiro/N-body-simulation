#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#include <iostream>

#include "constants.h"

__global__ void sim_kernel(double *pos, double *vel, double *acc, double *mas){
    cooperative_groups::grid_group g = cooperative_groups::this_grid();
    int particle_id = blockIdx.x *blockDim.x + threadIdx.x;

    for (int t = 0; t < CUDA_TIME_LENGTH; t++) {
        // Update pos[t+1]
        pos[((t + 1) % CUDA_TIME_LENGTH) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] =
            pos[(t % CUDA_TIME_LENGTH) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] +
            vel[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] * dt +
            0.5 * acc[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] * dt *
                dt;
        pos[((t + 1) % CUDA_TIME_LENGTH) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] =
            pos[(t % CUDA_TIME_LENGTH) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] +
            vel[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] * dt +
            0.5 * acc[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] * dt *
                dt;

        g.sync();

        // Update acc[t+1]
        double acc_x = 0, acc_y = 0;
        for (int i = 0; i < N_PARTICLE; i++) {
            if (i == particle_id) continue;
            double dx =
                pos[((t + 1) % CUDA_TIME_LENGTH) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] -
                pos[((t + 1) % CUDA_TIME_LENGTH) * N_PARTICLE * DIMENSION + i * DIMENSION + 0];
            double dy =
                pos[((t + 1) % CUDA_TIME_LENGTH) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] -
                pos[((t + 1) % CUDA_TIME_LENGTH) * N_PARTICLE * DIMENSION + i * DIMENSION + 1];
            double r = sqrt(dx * dx + dy * dy) + POS_EPS;
            acc_x += -G * mas[i] * dx / (r * r * r);
            acc_y += -G * mas[i] * dy / (r * r * r);
        }
        acc[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id  * DIMENSION + 0] = acc_x;
        acc[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id  * DIMENSION + 1] = acc_y;

        g.sync();

        // Update vel[t+1]
        vel[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] =
            vel[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] +
            0.5 *
                (acc[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] +
                 acc[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION +
                     0]) *
                dt;
        vel[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] =
            vel[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] +
            0.5 *
                (acc[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] +
                 acc[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION +
                     1]) *
                dt;
    }
}

void call_cuda_sim(double *pos_host, double *vel_host, double *acc_host, double *mas_host) {
    double *pos, *mas, *acc, *vel;
    size_t pos_size = sizeof(double) * CUDA_TIME_LENGTH * N_PARTICLE * DIMENSION;
    size_t mas_size = sizeof(double) * N_PARTICLE;
    size_t vel_size = sizeof(double) * N_PARTICLE * 2 * DIMENSION;
    size_t acc_size = sizeof(double) * N_PARTICLE * 2 * DIMENSION;

    hipError_t err;

    err = hipMalloc(&pos, pos_size);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipMalloc(&mas, mas_size);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipMalloc(&vel, vel_size);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipMalloc(&acc, acc_size);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }


    err = hipMemcpy(pos, pos_host, pos_size, hipMemcpyHostToDevice);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipMemcpy(mas, mas_host, mas_size, hipMemcpyHostToDevice);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipMemcpy(vel, vel_host, vel_size, hipMemcpyHostToDevice);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipMemcpy(acc, acc_host, acc_size, hipMemcpyHostToDevice);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }


    const void* args[]= {&pos, &vel, &acc, &mas};
    dim3 grid(N_PARTICLE / N_THREAD_PER_BLOCK, 1, 1);
	dim3 block(N_THREAD_PER_BLOCK, 1, 1);
    
    for(int t = 0;t < TIME_LENGTH / CUDA_TIME_LENGTH; ++t){
	    err = hipLaunchCooperativeKernel((void*)&sim_kernel, grid, block, (void**)args);
        if(err != hipSuccess){
            std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
        }
        err = hipMemcpy((char*)pos_host + pos_size * t + sizeof(double) * N_PARTICLE * DIMENSION, (char*)pos + sizeof(double) * N_PARTICLE * DIMENSION,
                sizeof(double) * (CUDA_TIME_LENGTH - 1) * N_PARTICLE * DIMENSION, hipMemcpyDeviceToHost);
        if(err != hipSuccess){
            std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
        }
        if(t < TIME_LENGTH / CUDA_TIME_LENGTH - 1){
            err = hipMemcpy((char*)pos_host + pos_size * (t + 1), pos,
                    sizeof(double) * N_PARTICLE * DIMENSION, hipMemcpyDeviceToHost);
            if(err != hipSuccess){
                std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
            }
        }
        std::cout << t << " / " << TIME_LENGTH / CUDA_TIME_LENGTH << " has finished." << std::endl;
    }

    err = hipFree(pos);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipFree(mas);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipFree(acc);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
    err = hipFree(vel);
    if(err != hipSuccess){
        std::cerr << "GPUassert: " << hipGetErrorString(err) << " " << __FILE__ << " " << __LINE__ << std::endl;
    }
}
