#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#include "constants.h"

__global__ void sim_kernel(double *pos, double *vel, double *acc, double *mas){
    cooperative_groups::grid_group g = cooperative_groups::this_grid();
    int particle_id = blockIdx.x *blockDim.x + threadIdx.x;

    for (int t = 0; t < TIME_LENGTH - 1; t++) {
        // Update pos[t+1]
            pos[(t + 1) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] =
                pos[t * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] +
                vel[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] * dt +
                0.5 * acc[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] * dt *
                    dt;
            pos[(t + 1) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] =
                pos[t * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] +
                vel[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] * dt +
                0.5 * acc[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] * dt *
                    dt;

        g.sync();

        // Update acc[t+1]
        double acc_x = 0, acc_y = 0;
        for (int i = 0; i < N_PARTICLE; i++) {
            if (i == particle_id) continue;
            double dx =
                pos[(t + 1) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] -
                pos[(t + 1) * N_PARTICLE * DIMENSION + i * DIMENSION + 0];
            double dy =
                pos[(t + 1) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] -
                pos[(t + 1) * N_PARTICLE * DIMENSION + i * DIMENSION + 1];
            double r = sqrt(dx * dx + dy * dy);
            if (r < POS_EPS) r = POS_EPS;
            acc_x += -G * mas[i] * dx / (r * r * r);
            acc_y += -G * mas[i] * dy / (r * r * r);
        }
        acc[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id  * DIMENSION + 0] = acc_x;
        acc[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id  * DIMENSION + 1] = acc_y;

        g.sync();

        // Update vel[t+1]
        vel[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] =
            vel[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] +
            0.5 *
                (acc[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 0] +
                 acc[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION +
                     0]) *
                dt;
        vel[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] =
            vel[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] +
            0.5 *
                (acc[(t % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION + 1] +
                 acc[((t + 1) % 2) * N_PARTICLE * DIMENSION + particle_id * DIMENSION +
                     1]) *
                dt;
    }
}

void call_cuda_sim(double *pos_host, double *vel_host, double *acc_host, double *mas_host) {
    double *pos, *mas, *acc, *vel;
    size_t pos_size = sizeof(double) * N_PARTICLE * TIME_LENGTH *
                           DIMENSION;
    size_t mas_size = sizeof(double) * N_PARTICLE;
    size_t vel_size = sizeof(double) * N_PARTICLE * 2 * DIMENSION;
    size_t acc_size = sizeof(double) * N_PARTICLE * 2 * DIMENSION;

    hipMalloc(&pos, pos_size);
    hipMalloc(&mas, mas_size);
    hipMalloc(&vel, vel_size);
    hipMalloc(&acc, acc_size);

    hipMemcpy(pos, pos_host, pos_size, hipMemcpyHostToDevice);
    hipMemcpy(mas, mas_host, mas_size, hipMemcpyHostToDevice);
    hipMemcpy(vel, vel_host, vel_size, hipMemcpyHostToDevice);
    hipMemcpy(acc, acc_host, acc_size, hipMemcpyHostToDevice);


    const void* args[]= {&pos, &vel, &acc, &mas};
    dim3 grid(N_PARTICLE / N_THREAD_PER_BLOCK, 1, 1);
	dim3 block(N_THREAD_PER_BLOCK, 1, 1);
	hipLaunchCooperativeKernel((void*)&sim_kernel, grid, block, (void**)args);

    hipMemcpy(pos_host, pos, pos_size, hipMemcpyDeviceToHost);

    hipFree(pos);
    hipFree(mas);
    hipFree(acc);
    hipFree(vel);
}
